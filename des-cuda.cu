#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <cstdio>
#include <string>
#include <random>

#include "des.h"
#include "utils.h"
#include "cuda_utils.h"
#include "des_kernel.h"

using namespace std;

void parse_args(int argc, char **argv, int *key_length);

void usage(char *name);

void parse_args(int argc, char **argv, int *key_length) {
    if (argc < 2) {
        usage(argv[0]);
    }
    *key_length = atoi(argv[1]);
    if (*key_length <= 0 || *key_length > 64) {
        usage(argv[0]);
    }
}

void usage(char *name) {
    printf("Usage:\n %s key_length(1-64)\n", name);
    exit(EXIT_FAILURE);
}


int main(int argc, char **argv) {

    int numberOfPasswords = 1 << 20;

    string *passwordsList = new string[numberOfPasswords];
    generatePasswords(numberOfPasswords, passwordsList);

    int key_length;
    parse_args(argc, argv, &key_length);
    printf("Key length: %d \n", key_length);

    random_device rd;
    mt19937 gen(rd());
    uniform_int_distribution<> distrib(0, (numberOfPasswords) - 1);

    string selectedPassword = passwordsList[distrib(gen)];

    char _selectedPassword[9];
    for (int i = 0; i < selectedPassword.size(); i++)
        _selectedPassword[i] = selectedPassword[i];
    _selectedPassword[8] = '\0';

    printf("Password to be hacked: %s\n", _selectedPassword);

    uint64_t passwordKey = *(uint64_t *) _selectedPassword;
    uint64_t encodedPassword = full_des_encode_block(passwordKey, passwordKey);

    /* START CRACKING */
    _cudaSetDevice(0);
    hipMemcpyToSymbol(HIP_SYMBOL(devEncodedPassword), &encodedPassword, sizeof(uint64_t));
    hipMemcpyToSymbol(HIP_SYMBOL(passwordsListSize), &numberOfPasswords, sizeof(int));

    char *devPasswordsList;
    _cudaMalloc((void **) &devPasswordsList, (numberOfPasswords) * 8 * sizeof(char));
    for (int i = 0; i < numberOfPasswords; i++)
        _cudaMemcpy(&devPasswordsList[i * 8], passwordsList[i].c_str(), 8 * sizeof(char), hipMemcpyHostToDevice);

    int *devFoundFlag;
    _cudaMalloc((void **) &devFoundFlag, sizeof(int));
    _cudaMemset(devFoundFlag, 0, sizeof(int));

    char *devResult;
    _cudaMalloc((void **) &devResult, 9 * sizeof(char));

    dim3 dimGrid = 1 << 7; //2^7
    dim3 dimBlock = 1 << 9; //2^9

    clock_t start = clock();

    cudaHackPassword<<<dimGrid, dimBlock>>>(devPasswordsList, devFoundFlag, devResult);
    _cudaDeviceSynchronize("cudaHackPassword");

    clock_t end = clock();

    char foundPassword[9];
    foundPassword[8] = '\0';

    _cudaMemcpy(&foundPassword, devResult, 8 * sizeof(char), hipMemcpyDeviceToHost);
    hipFree(devFoundFlag);
    hipFree(devResult);
    hipFree(devPasswordsList);

    float seconds = (float) (end - start) / CLOCKS_PER_SEC;
    printf("Found password: %s, seconds: %f\n", foundPassword, seconds);

    return EXIT_SUCCESS;
}