#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <cstdio>
#include <string>
#include <random>

#include "des.h"
#include "utils.h"
#include "cuda_utils.h"
#include "des_kernel.h"

using namespace std;

void parse_args(int argc, char **argv, int *key_length);

void usage(char *name);

void parse_args(int argc, char **argv, int *key_length) {
    if (argc < 2) {
        usage(argv[0]);
    }
    *key_length = atoi(argv[1]);
    if (*key_length <= 0 || *key_length > 64) {
        usage(argv[0]);
    }
}

void usage(char *name) {
    printf("Usage:\n %s key_length(1-64)\n", name);
    exit(EXIT_FAILURE);
}


int main(int argc, char **argv) {

    int key_length;
    parse_args(argc, argv, &key_length);

    /* PASSWORDS GENERATION */

    unsigned int numberOfPasswords = 1 << 27; // 2^27

    char *passwordsList = new char[8 * numberOfPasswords];
    generatePasswords(numberOfPasswords, passwordsList);

    /* PASSWORD SELECTION */

    random_device rd;
    mt19937 gen(rd());
    uniform_int_distribution<> distrib(0, (numberOfPasswords) - 1);
    unsigned int randomIndex = distrib(gen) * 8;

    char *selectedPassword = &passwordsList[randomIndex];

    char _selectedPassword[9];
    for (int i = 0; i < 8; i++)
        _selectedPassword[i] = selectedPassword[i];
    _selectedPassword[8] = '\0';

    printf("Password to be hacked: %s\n", _selectedPassword);

    uint64_t passwordKey = *(uint64_t *) _selectedPassword;
    uint64_t encodedPassword = full_des_encode_block(passwordKey, passwordKey);

    /* START CRACKING */

    _cudaSetDevice(0);
    hipMemcpyToSymbol(HIP_SYMBOL(devEncodedPassword), &encodedPassword, sizeof(uint64_t));
    hipMemcpyToSymbol(HIP_SYMBOL(passwordsListSize), &numberOfPasswords, sizeof(unsigned int));

    char *devPasswordsList;
    _cudaMalloc((void **) &devPasswordsList, (numberOfPasswords) * 8 * sizeof(char));
    _cudaMemcpy(devPasswordsList, passwordsList, (numberOfPasswords) * 8 * sizeof(char), hipMemcpyHostToDevice);

    int *devFoundFlag;
    _cudaMalloc((void **) &devFoundFlag, sizeof(int));
    _cudaMemset(devFoundFlag, 0, sizeof(int));

    char *devResult;
    _cudaMalloc((void **) &devResult, 9 * sizeof(char));

    dim3 dimGrid = 1 << 7; // 2^7
    dim3 dimBlock = 1 << 9; // 2^9

    clock_t start = clock();

    cudaHackPassword<<<dimGrid, dimBlock>>>(devPasswordsList, devFoundFlag, devResult);
    _cudaDeviceSynchronize("cudaHackPassword");

    clock_t end = clock();

    char foundPassword[9];
    foundPassword[8] = '\0';

    _cudaMemcpy(foundPassword, devResult, 8 * sizeof(char), hipMemcpyDeviceToHost);
    hipFree(devFoundFlag);
    hipFree(devResult);
    hipFree(devPasswordsList);
    delete[] passwordsList;

    float seconds = (float) (end - start) / CLOCKS_PER_SEC;
    printf("Found password: %s in %f seconds\n"
           "Total number of passwords were: %d\n"
           "Cracked password had index: %d", foundPassword, seconds, numberOfPasswords, randomIndex / 8);

    return EXIT_SUCCESS;
}