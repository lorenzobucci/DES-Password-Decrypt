#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <cstdio>

#include "des.h"
#include "utils.h"
#include "cuda_utils.h"
#include "des_kernel.h"

void parse_args(int argc, char **argv, int *key_length);

void usage(char *name);

void parse_args(int argc, char **argv, int *key_length) {
    if (argc < 2) {
        usage(argv[0]);
    }
    *key_length = atoi(argv[1]);
    if (*key_length <= 0 || *key_length > 64) {
        usage(argv[0]);
    }
}

void usage(char *name) {
    printf("Usage:\n %s key_length(1-64)\n", name);
    exit(EXIT_FAILURE);
}

int main(int argc, char **argv) {
    //generatePasswords(100, "plaintextPasswords.txt");
    int key_length;
    parse_args(argc, argv, &key_length);
    printf("Key length: %d \n", key_length);

    FILE *fPtr;
    fPtr = fopen("plaintextPasswords.txt", "r");
    if (fPtr == nullptr) {
        printf("Unable to read file!");
        exit(EXIT_FAILURE);
    }

    char plaintextPassword[8];
    fscanf(fPtr, "%s", plaintextPassword);
    fclose(fPtr);

    uint64_t passwordKey = *(uint64_t *) plaintextPassword;;
    uint64_t encodedPassword = full_des_encode_block(passwordKey, passwordKey);

    /* START CRACKING */
    _cudaSetDevice(0);
    hipMemcpyToSymbol(HIP_SYMBOL(devEncodedPassword), &encodedPassword, sizeof(uint64_t));

    int *foundFlag;
    _cudaMalloc((void **) &foundFlag, sizeof(int));
    _cudaMemset(foundFlag, 0, sizeof(int));

    uint64_t *devResult;
    _cudaMalloc((void **) &devResult, sizeof(uint64_t));

    dim3 dimGrid = 1 << 21; //2^
    dim3 dimBlock = 1 << 10; //2^

    clock_t start = clock();

    cudaHackPassword<<<dimGrid, dimBlock>>>(foundFlag, devResult);
    _cudaDeviceSynchronize("cudaHackPassword");

    clock_t end = clock();

    uint64_t foundKey;

    _cudaMemcpy(&foundKey, devResult, sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipFree(foundFlag);
    hipFree(devResult);

    float seconds = (float) (end - start) / CLOCKS_PER_SEC;
    printf("key length: %d, seconds: %f\n", key_length, seconds);

    //bits_print_grouped(encoded,8,64);
    //bits_print_grouped(full_des_encode_block(cracked_key,block),8,64);
    return EXIT_SUCCESS;
}