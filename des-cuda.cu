#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <cstdio>

#include "des.h"
#include "utils.h"
#include "cuda_utils.h"
#include "des_kernel.h"

void parse_args(int argc, char **argv, int *key_length);

void usage(char *name);

void parse_args(int argc, char **argv, int *key_length) {
    if (argc < 2) {
        usage(argv[0]);
    }
    *key_length = atoi(argv[1]);
    if (*key_length <= 0 || *key_length > 64) {
        usage(argv[0]);
    }
}

void usage(char *name) {
    printf("Usage:\n %s key_length(1-64)\n", name);
    exit(EXIT_FAILURE);
}

int main(int argc, char **argv) {
    //generatePasswords(100, "plaintextPasswords.txt");
    int key_length;
    parse_args(argc, argv, &key_length);
    printf("Key length: %d \n", key_length);

    FILE *fPtr;
    fPtr = fopen("plaintextPasswords.txt", "r");
    if (fPtr == nullptr) {
        printf("Unable to read file!");
        exit(EXIT_FAILURE);
    }

    char plaintextPassword[8];
    fscanf(fPtr, "%s", plaintextPassword);
    fclose(fPtr);

    uint64_t passwordKey = *(uint64_t *) plaintextPassword;;
    uint64_t encodedPassword = full_des_encode_block(passwordKey, passwordKey);

    /* START CRACKING */
    _cudaSetDevice(0);

    clock_t start = clock();

    uint64_t crackedKey = 0;
    uint64_t encodedCrackedKey = 0;
    run_des_encode_block(crackedKey, crackedKey, &encodedCrackedKey);
    while (encodedCrackedKey != encodedPassword) {
        crackedKey++;
        encodedCrackedKey = full_des_encode_block(crackedKey, crackedKey);
        run_des_encode_block(crackedKey, crackedKey, &encodedCrackedKey);
    }

    clock_t end = clock();
    float seconds = (float) (end - start) / CLOCKS_PER_SEC;
    printf("key length: %d, seconds: %f\n", key_length, seconds);

    //bits_print_grouped(encoded,8,64);
    //bits_print_grouped(full_des_encode_block(cracked_key,block),8,64);
    return EXIT_SUCCESS;
}