#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <cstdio>
#include <string>
#include <array>

#include "des.h"
#include "utils.h"
#include "cuda_utils.h"
#include "des_kernel.h"

using namespace std;

void parse_args(int argc, char **argv, int *key_length);

void usage(char *name);

void cudaCracking(unsigned int numberOfPasswords, const char *passwordsList, uint64_t encodedPassword);

void cpuCracking(unsigned int numberOfPasswords, const char *passwordsList, uint64_t encodedPassword);

void parse_args(int argc, char **argv, int *key_length) {
    if (argc < 2) {
        usage(argv[0]);
    }
    *key_length = atoi(argv[1]);
    if (*key_length <= 0 || *key_length > 64) {
        usage(argv[0]);
    }
}

void usage(char *name) {
    printf("Usage:\n %s key_length(1-64)\n", name);
    exit(EXIT_FAILURE);
}


int main(int argc, char **argv) {

    int key_length;
    parse_args(argc, argv, &key_length);

    /* PASSWORDS GENERATION */

    unsigned int numberOfPasswords = 1 << 20; // 2^20

    printf("Generating %d passwords...\n", numberOfPasswords);

    char *passwordsList = new char[8 * numberOfPasswords];
    generatePasswords(numberOfPasswords, passwordsList);

    /* PASSWORD SELECTION */

    array<unsigned int, 3> passwordIndexes = {0, numberOfPasswords / 2, numberOfPasswords - 1};

    for (unsigned int passwordIndex : passwordIndexes) {

        char *selectedPassword = &passwordsList[passwordIndex * 8];

        char _selectedPassword[9];
        for (int i = 0; i < 8; i++)
            _selectedPassword[i] = selectedPassword[i];
        _selectedPassword[8] = '\0';


        printf("\nPassword to be hacked: %s, with index %d\n\n", _selectedPassword, passwordIndex);

        uint64_t passwordKey = *(uint64_t *) _selectedPassword;
        uint64_t encodedPassword = full_des_encode_block(passwordKey, passwordKey);

        /* START CRACKING */

        printf("Trying to hack using CPU\n\n");
        for (int attempt = 0; attempt < 2; attempt++) {
            printf("Attempt %d of 2\n", attempt + 1);
            cpuCracking(numberOfPasswords, passwordsList, encodedPassword);
        }

        printf("\nTrying to hack using GPU\n\n");
        for (int attempt = 0; attempt < 2; attempt++) {
            printf("Attempt %d of 2\n", attempt + 1);
            cudaCracking(numberOfPasswords, passwordsList, encodedPassword);
        }

    }

    delete[] passwordsList;

    return EXIT_SUCCESS;
}

void cudaCracking(unsigned int numberOfPasswords, const char *passwordsList, uint64_t encodedPassword) {
    _cudaSetDevice(0);
    hipMemcpyToSymbol(HIP_SYMBOL(devEncodedPassword), &encodedPassword, sizeof(uint64_t));
    hipMemcpyToSymbol(HIP_SYMBOL(passwordsListSize), &numberOfPasswords, sizeof(unsigned int));

    char *devPasswordsList;
    _cudaMalloc((void **) &devPasswordsList, (numberOfPasswords) * 8 * sizeof(char));
    _cudaMemcpy(devPasswordsList, passwordsList, (numberOfPasswords) * 8 * sizeof(char), hipMemcpyHostToDevice);

    int *devFoundFlag;
    _cudaMalloc((void **) &devFoundFlag, sizeof(int));
    _cudaMemset(devFoundFlag, 0, sizeof(int));

    char *devResult;
    _cudaMalloc((void **) &devResult, 9 * sizeof(char));

    dim3 dimGrid = 1 << 7; // 2^7
    dim3 dimBlock = 1 << 9; // 2^9

    clock_t start = clock();

    cudaHackPassword<<<dimGrid, dimBlock>>>(devPasswordsList, devFoundFlag, devResult);
    _cudaDeviceSynchronize("cudaHackPassword");

    clock_t end = clock();

    char foundPassword[9];
    foundPassword[8] = '\0';

    _cudaMemcpy(foundPassword, devResult, 8 * sizeof(char), hipMemcpyDeviceToHost);
    hipFree(devFoundFlag);
    hipFree(devResult);
    hipFree(devPasswordsList);

    float seconds = (float) (end - start) / CLOCKS_PER_SEC;
    printf("Found password: %s in %f seconds\n", foundPassword, seconds);
}

void cpuCracking(unsigned int numberOfPasswords, const char *passwordsList, uint64_t encodedPassword) {
    uint64_t encodedCrackedKey = 0;
    char crackedPassword[8];

    clock_t start = clock();

    for (unsigned int index = 0; index < numberOfPasswords && encodedCrackedKey != encodedPassword; index++) {
        for (int i = 0; i < 8; i++)
            crackedPassword[i] = passwordsList[8 * index + i];

        uint64_t crackedKey = *(uint64_t *) crackedPassword;
        encodedCrackedKey = full_des_encode_block(crackedKey, crackedKey);
    }

    clock_t end = clock();

    char foundPassword[9];
    for (int i = 0; i < 8; i++)
        foundPassword[i] = crackedPassword[i];
    foundPassword[8] = '\0';

    float seconds = (float) (end - start) / CLOCKS_PER_SEC;
    printf("Found password: %s in %f seconds\n", foundPassword, seconds);

}
