#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <cstdio>
#include <chrono>
#include <string>
#include <array>

#include "des.h"
#include "utils.h"
#include "cuda_utils.h"
#include "des_kernel.h"

using namespace std;

void cudaCracking(unsigned int numberOfPasswords, const char *passwordsList, uint64_t encodedPassword);

void cpuCracking(unsigned int numberOfPasswords, const char *passwordsList, uint64_t encodedPassword);


int main() {

    /* PASSWORDS GENERATION */

    unsigned int numberOfPasswords = 1 << 22; // 2^20

    printf("Generating %d passwords...\n", numberOfPasswords);

    char *passwordsList = new char[8 * numberOfPasswords];
    generatePasswords(numberOfPasswords, passwordsList);

    /* PASSWORD SELECTION */

    array<unsigned int, 3> passwordIndexes = {0, numberOfPasswords / 2, numberOfPasswords - 1};

    for (unsigned int passwordIndex : passwordIndexes) {

        char *selectedPassword = &passwordsList[passwordIndex * 8];

        char _selectedPassword[9];
        for (int i = 0; i < 8; i++)
            _selectedPassword[i] = selectedPassword[i];
        _selectedPassword[8] = '\0';


        printf("\nPassword to be hacked: %s, with index %d\n\n", _selectedPassword, passwordIndex);

        uint64_t passwordKey = *(uint64_t *) _selectedPassword;
        uint64_t encodedPassword = full_des_encode_block(passwordKey, passwordKey);

        /* START CRACKING */

        printf("Trying to hack using CPU\n\n");
        for (int attempt = 0; attempt < 2; attempt++) {
            printf("Attempt %d of 2\n", attempt + 1);
            cpuCracking(numberOfPasswords, passwordsList, encodedPassword);
        }

        printf("\nTrying to hack using GPU\n\n");
        for (int attempt = 0; attempt < 2; attempt++) {
            printf("Attempt %d of 2\n", attempt + 1);
            cudaCracking(numberOfPasswords, passwordsList, encodedPassword);
        }

    }

    delete[] passwordsList;

    return EXIT_SUCCESS;
}

void cudaCracking(unsigned int numberOfPasswords, const char *passwordsList, uint64_t encodedPassword) {
    _cudaSetDevice(0);
    hipMemcpyToSymbol(HIP_SYMBOL(devEncodedPassword), &encodedPassword, sizeof(uint64_t));
    hipMemcpyToSymbol(HIP_SYMBOL(passwordsListSize), &numberOfPasswords, sizeof(unsigned int));

    char *devPasswordsList;
    _cudaMalloc((void **) &devPasswordsList, (numberOfPasswords) * 8 * sizeof(char));
    _cudaMemcpy(devPasswordsList, passwordsList, (numberOfPasswords) * 8 * sizeof(char), hipMemcpyHostToDevice);

    int *devFoundFlag;
    _cudaMalloc((void **) &devFoundFlag, sizeof(int));
    _cudaMemset(devFoundFlag, 0, sizeof(int));

    char *devResult;
    _cudaMalloc((void **) &devResult, 9 * sizeof(char));

    dim3 dimGrid = 2048;
    dim3 dimBlock = 512;

    auto start = chrono::high_resolution_clock::now();

    cudaHackPassword<<<dimGrid, dimBlock>>>(devPasswordsList, devFoundFlag, devResult);
    _cudaDeviceSynchronize("cudaHackPassword");

    auto end = chrono::high_resolution_clock::now();

    char foundPassword[9];
    foundPassword[8] = '\0';

    _cudaMemcpy(foundPassword, devResult, 8 * sizeof(char), hipMemcpyDeviceToHost);
    hipFree(devFoundFlag);
    hipFree(devResult);
    hipFree(devPasswordsList);

    chrono::duration<float> diff = end - start;
    printf("Found password: %s in %f seconds\n", foundPassword, diff.count());
}

void cpuCracking(unsigned int numberOfPasswords, const char *passwordsList, uint64_t encodedPassword) {
    uint64_t encodedCrackedKey = 0;
    char crackedPassword[8];

    auto start = chrono::high_resolution_clock::now();

    for (unsigned int index = 0; index < numberOfPasswords && encodedCrackedKey != encodedPassword; index++) {
        for (int i = 0; i < 8; i++)
            crackedPassword[i] = passwordsList[8 * index + i];

        uint64_t crackedKey = *(uint64_t *) crackedPassword;
        encodedCrackedKey = full_des_encode_block(crackedKey, crackedKey);
    }

    auto end = chrono::high_resolution_clock::now();

    char foundPassword[9];
    for (int i = 0; i < 8; i++)
        foundPassword[i] = crackedPassword[i];
    foundPassword[8] = '\0';

    chrono::duration<float> diff = end - start;
    printf("Found password: %s in %f seconds\n", foundPassword, diff.count());

}
