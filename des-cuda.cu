#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>

#include "c_utils.h"
#include "des.h"
#include "des_utils.h"
#include "bit_utils.h"
#include "des_consts.h"
#include "des_kernel.h"
#include "cuda_utils.h"

void parse_args(int argc, char **argv, int *key_length);

void usage(char *name);

void parse_args(int argc, char **argv, int *key_length) {
    if (argc < 2) {
        usage(argv[0]);
    }
    *key_length = atoi(argv[1]);
    if (*key_length <= 0 || *key_length > 64) {
        usage(argv[0]);
    }
}

void usage(char *name) {
    printf("Usage:\n %s key_length(1-64)\n", name);
    exit(EXIT_FAILURE);
}

int main(int argc, char **argv) {
    int key_length;
    parse_args(argc, argv, &key_length);
    printf("Key length: %d \n", key_length);
    uint64_t key = des_generate_key();
    uint64_t block = key; //0x0123456789ABCDEF;
    uint64_t encoded = full_des_encode_block(key, block);

    //_cudaSetDevice(0);

    printf("Real key:\n");
    bits_print_grouped(key, 8, 64);
    printf("Encoded block:\n");
    bits_print_grouped(encoded, 8, 64);
    printf("Cracking...\n");
    uint64_t cracked_key = key - 5;

    clock_t start = clock();
    for (int i = 0; i < 10; i++) {
        uint64_t decrypted_block = 0;
        //run_des_encode_block(cracked_key, block, &decrypted_block);
        decrypted_block = full_des_encode_block(cracked_key, cracked_key);
        if (decrypted_block == encoded) {
            printf("Found !! iteration: %d\n", i);
            printf("Cracked key:\n");
            bits_print_grouped(cracked_key, 8, 64);
            printf("Cracked block:\n");
            bits_print_grouped(decrypted_block, 8, 64);
        }
        cracked_key++;
    }

    clock_t end = clock();
    float seconds = (float) (end - start) / CLOCKS_PER_SEC;
    printf("key length: %d, seconds: %f\n", key_length, seconds);

    //bits_print_grouped(encoded,8,64);
    //bits_print_grouped(full_des_encode_block(cracked_key,block),8,64);
    return EXIT_SUCCESS;
}
